
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32  // blocks
#define M 4   // threads per block

__global__ void mykernel(void) {
}

__global__ void add(int *a, int *b, int *c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n)
    c[index] = a[index] + b[index];
}


void random_ints(int* x, int size) {
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%10;
	}
}

int main(void) {

  int *a, *b, *c;       // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c

  int size =  N * sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with N blocks and M threads per block
  add<<<N/M,M>>>(d_a, d_b, d_c, N);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for (int i=0;i<N;i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;

}
