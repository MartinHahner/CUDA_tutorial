
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32

__global__ void mykernel(void) {
}

__global__ void add(int *a, int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}


void random_ints(int* x, int size) {
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%10;
	}
}

int main(void) {

  int *a, *b, *c;       // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c

  int size =  N * sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  add<<<N,1>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for (int i=0;i<N;i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;

}
